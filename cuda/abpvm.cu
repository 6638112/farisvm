#include "hip/hip_runtime.h"
#include "abpvm.hpp"

#include <algorithm>
#include <iostream>
#include <sstream>
#include <regex>

#include <stdio.h>

#define INST_MAX 4096

#define OP_CHAR        0x00
#define OP_SKIP_TO     0x80
#define OP_SKIP_SCHEME 0x83
#define OP_MATCH       0x84

#define CHAR_TAIL      0
#define CHAR_HEAD      1
#define CHAR_SEPARATOR 2

#define IS_OP_CHAR(INST) (!(0x80 & (INST)))
#define IS_OP_SKIP_SCHEME(INST) ((char)0x83 == (INST))
#define IS_OP_MATCH(INST) ((char)0x84 == (INST))

#define TO_LOWER(CH_) (('A' <= (CH_) && (CH_) <= 'Z') ? (CH_) + ('a' - 'A') : (CH_))
#define UNSIGNED(CH_) (int)(unsigned char)(CH_)

#define MAX_BLOCK_DIM 4096
#define MIN_BLOCK_DIM 32

#define SHM_SIZE 49152

#define MAX_CODE_SIZE (SHM_SIZE / MAX_BLOCK_DIM)

#define MAX_QUERY_LEN (1024 * 32)
#define MAX_QUERY_NUM 100

#define MAX_RESULT 64

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void
gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
                file, line);
        if (abort) exit(code);
    }
}

// characters for URL by RFC 3986
int urlchar[256] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,
                    1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 1, 0, 1,
                    1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,
                    1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 1, 0, 1,
                    0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,
                    1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 1, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

// separators
int sepchar[256] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    1, 1, 1, 1, 1, 0, 1, 1, 1, 1, 1, 1, 1, 0, 0, 1,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1,
                    1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 0,
                    1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

int schemechar[256] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                       0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                       0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 1, 1, 0,
                       1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0,
                       0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,
                       1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0,
                       0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,
                       1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0,
                       0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                       0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                       0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                       0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                       0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                       0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                       0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                       0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

__constant__ int  d_urlchar[256];
__constant__ int  d_sepchar[256];
__constant__ int  d_schemechar[256];

// Beginning of GPU Architecture definitions
inline int
_ConvertSMVer2Cores(int major, int minor)
{
    // Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
    typedef struct
    {
        int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
        int Cores;
    } sSMtoCores;

    sSMtoCores nGpuArchCoresPerSM[] =
    {
        { 0x20, 32 }, // Fermi Generation (SM 2.0) GF100 class
        { 0x21, 48 }, // Fermi Generation (SM 2.1) GF10x class
        { 0x30, 192}, // Kepler Generation (SM 3.0) GK10x class
        { 0x32, 192}, // Kepler Generation (SM 3.2) GK10x class
        { 0x35, 192}, // Kepler Generation (SM 3.5) GK11x class
        { 0x37, 192}, // Kepler Generation (SM 3.7) GK21x class
        { 0x50, 128}, // Maxwell Generation (SM 5.0) GM10x class
        { 0x52, 128}, // Maxwell Generation (SM 5.2) GM20x class
        {   -1, -1 }
    };

    int index = 0;

    while (nGpuArchCoresPerSM[index].SM != -1)
    {
        if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor))
        {
            return nGpuArchCoresPerSM[index].Cores;
        }

        index++;
    }

    // If we don't find the values, we default use the previous one to run properly
    printf("MapSMtoCores for SM %d.%d is undefined.  Default to use %d Cores/SM\n", major, minor, nGpuArchCoresPerSM[index-1].Cores);
    return nGpuArchCoresPerSM[index-1].Cores;
}

__device__
void
gpu_print_asm(char *code, int num_inst)
{
    for (int i = 0; i < num_inst; i++) {
        if (IS_OP_CHAR(*code)) {
            printf("char ");
            if (*code == CHAR_HEAD) {
                printf("head\n");
            } else if (*code == CHAR_TAIL) {
                printf("tail\n");
            } else if (*code == CHAR_SEPARATOR) {
                printf("separator\n");
            } else {
                printf("%c\n", *code);
            }
        } else if (IS_OP_MATCH(*code)) {
            printf("match\n");
            return;
        } else if (IS_OP_SKIP_SCHEME(*code)) {
            printf("skip_scheme\n");
        } else {
            char c = 0x7f & *code;
            printf("skip_to ");
            if (c == CHAR_HEAD) {
                printf("head\n");
            } else if (c == CHAR_TAIL) {
                printf("tail\n");
            } else if (c == CHAR_SEPARATOR) {
                printf("separator\n");
            } else {
                printf("%c\n", c);
            }
        }

        code++;
    }
}

__device__
bool
gpu_vmrun(char *pc, char *sp, int num_inst)
{
    for (int i = 0; i < num_inst; i++) {
        if (IS_OP_CHAR(*pc)) {
            if (*pc == CHAR_SEPARATOR) {
                if (! d_sepchar[(unsigned char)*sp]) {
                    return false;
                }
            } else {
                if (*pc != *sp) {
                    return false;
                }
            }
            sp++;
        } else if (IS_OP_MATCH(*pc)) {
            return true;
        } else {
            // skip_to
            char c = 0x7f & *pc;
            if (c == CHAR_SEPARATOR) {
                while (! d_sepchar[(unsigned char)*sp]) {
                    if (*sp == '\0') {
                        return false;
                    }
                    sp++;
                }
            } else {
                while (c != *sp) {
                    if (*sp == '\0') {
                        return false;
                    }
                    sp++;
                }
            }
        }

        pc++;
    }

    // never reach here
    return false;
}

__global__
void
gpu_match(char *codes, int *codes_idx, int num_codes, int *scheme_len,
          char *query, char *query_lower, int query_num, int *result)
{
    for (int i = 0; i < query_num; i++) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        while (idx < num_codes) {
            char *sp;
            char *pc = &codes[codes_idx[idx]];
            abpvm::abpvm_head *head = (abpvm::abpvm_head*)pc;

            pc += sizeof(*head);

            if (head->flags & FLAG_MATCH_CASE) {
                sp = &query[i * MAX_QUERY_LEN];
            } else {
                sp = &query_lower[i * MAX_QUERY_LEN];
            }

            bool ret;
            bool check_head = false;
            if (*pc == CHAR_HEAD) {
                check_head = true;
                pc++;
                if (IS_OP_SKIP_SCHEME(*pc)) {
                    pc++;
                    sp += scheme_len[i];
                }
            }

            while (*sp != '\0') {
                ret = gpu_vmrun(pc, sp, head->num_inst);

                if (check_head || ret) {
                    break;
                }
                sp++;
            }

            if (ret) {
                for (int j = 0; j < MAX_RESULT; j++) {
                    int n = MAX_RESULT * query_num + j;
                    atomicCAS(&result[n], -1, idx);
                    if (result[n] == idx) {
                        break;
                    }
                }
            }

            idx += gridDim.x * blockDim.x;
        }
    }
};

bool
code_cmp(const char *lhs, const char *rhs)
{
    abpvm::abpvm_head *lhead, *rhead;

    lhead = (abpvm::abpvm_head*)lhs;
    rhead = (abpvm::abpvm_head*)rhs;

    lhs += sizeof(*lhead);
    rhs += sizeof(*rhead);

    int len = lhead->num_inst < rhead->num_inst ? lhead->num_inst : rhead->num_inst;

    int ret = memcmp(lhs, rhs, len);
    if (ret < 0) {
        return true;
    } else if (ret > 0) {
        return false;
    } else {
        return lhead->num_inst < rhead->num_inst;
    }
}

abpvm_exception::abpvm_exception(const std::string msg) : m_msg(msg)
{

}

abpvm_exception::~abpvm_exception() throw()
{

}

const char*
abpvm_exception::what() const throw()
{
    return m_msg.c_str();
}

abpvm_query::abpvm_query()
{
    m_uri = new char[MAX_QUERY_LEN];
    m_uri_lower = new char[MAX_QUERY_LEN];
    //gpuErrchk(hipHostMalloc((void**)&m_uri, MAX_QUERY_LEN));
    //gpuErrchk(hipHostMalloc((void**)&m_uri_lower, MAX_QUERY_LEN));
}

abpvm_query::~abpvm_query()
{
    delete[] m_uri;
    delete[] m_uri_lower;
    //gpuErrchk(hipHostFree(m_uri));
    //gpuErrchk(hipHostFree(m_uri_lower));
}

void
abpvm_query::set_uri(const std::string &uri)
{
    int len;
    if (uri.size() + 1 > MAX_QUERY_LEN) {
        len = MAX_QUERY_LEN - 1;
    } else {
        len = uri.size();
    }

    m_len = len + 1;

    memcpy(m_uri, uri.c_str(), len);
    memcpy(m_uri_lower, uri.c_str(), len);

    m_uri[len] = '\0';
    m_uri_lower[len] = '\0';

    std::transform(m_uri_lower, m_uri_lower + len,
                   m_uri_lower, ::tolower);

    size_t colon = uri.find(":");
    if (colon == std::string::npos) {
        m_domain = "";
        return;
    }

    size_t begin = colon + 1;
    while (begin < uri.size() && uri.at(begin) == '/') {
        begin++;
    }

    if (begin >= uri.size()) {
        m_domain = "";
        return;
    }

    size_t end = begin + 1;
    while (end < uri.size() && uri.at(end) != '/') {
        end++;
    }

    m_domain = uri.substr(begin, end - begin);

    m_domain_lower = m_domain;
    std::transform(m_domain_lower.begin(), m_domain_lower.end(),
                   m_domain_lower.begin(), ::tolower);
}

abpvm::abpvm() : m_d_codes_buf(nullptr),
                 m_d_codes_idx(nullptr),
                 m_need_gpu_init(true),
                 m_grid_dim(32),
                 m_block_dim(256),
                 m_code_bytes(0)
{
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_urlchar), urlchar, sizeof(urlchar)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_sepchar), sepchar, sizeof(sepchar)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_schemechar), schemechar, sizeof(schemechar)));

    gpuErrchk(hipFuncSetCacheConfig(reinterpret_cast<const void*>(gpu_match), hipFuncCachePreferL1));

    gpuErrchk(hipMalloc((void**)&m_d_query, MAX_QUERY_LEN * MAX_QUERY_NUM));
    gpuErrchk(hipMalloc((void**)&m_d_query_lower, MAX_QUERY_LEN * MAX_QUERY_NUM));
    gpuErrchk(hipMalloc((void**)&m_d_scheme_len, MAX_QUERY_NUM * sizeof(m_d_scheme_len[0])));
    gpuErrchk(hipMalloc((void**)&m_d_result, MAX_QUERY_NUM * MAX_RESULT * sizeof(m_d_result[0])));

    gpuErrchk(hipHostMalloc((void**)&m_result_init, MAX_QUERY_NUM * MAX_RESULT * sizeof(m_result_init[0])));

    memset(m_result_init, -1, MAX_QUERY_NUM * MAX_RESULT * sizeof(m_result_init[0]));

    get_gpu_prop();
}

abpvm::~abpvm()
{
    for (auto &p: m_codes) {
        delete[] p->code;
    }

    if (m_d_codes_buf != nullptr) {
        gpuErrchk(hipFree(m_d_codes_buf));
    }

    if (m_d_codes_idx != nullptr) {
        gpuErrchk(hipFree(m_d_codes_idx))
    }

    gpuErrchk(hipFree(m_d_query));
    gpuErrchk(hipFree(m_d_query_lower));
    gpuErrchk(hipFree(m_d_scheme_len));
    gpuErrchk(hipFree(m_d_result));

    gpuErrchk(hipHostFree(m_result_init));
}

void
abpvm::get_gpu_prop()
{
    int deviceCount = 0;
    gpuErrchk(hipGetDeviceCount(&deviceCount));

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    m_grid_dim = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount;
}

void
abpvm::init_gpu()
{
    if (m_need_gpu_init) {
        if (m_d_codes_buf != nullptr) {
            gpuErrchk(hipFree(m_d_codes_buf));
        }

        if (m_d_codes_idx != nullptr) {
            gpuErrchk(hipFree(m_d_codes_idx));
        }

        std::sort(m_codes.begin(), m_codes.end(),
                  [](const std::shared_ptr<abpvm_code> &lhs,
                     const std::shared_ptr<abpvm_code> &rhs)
                     {
                         abpvm_head *rhead, *lhead;
                         char *rc, *lc;

                         rhead = (abpvm_head*)lhs->code;
                         lhead = (abpvm_head*)rhs->code;

                         rc = lhs->code + sizeof(rhead);
                         lc = lhs->code + sizeof(lhead);

                         int len = (lhead->num_inst < rhead->num_inst) ? lhead->num_inst : rhead->num_inst;
                         return memcmp(lc, rc, len);
                     });

        int num_codes = m_codes.size();

        char *codes = new char[m_code_bytes];
        int  *idx   = new int[m_codes.size()];
        int j = 0;
        int pos = 0;
        for (auto &code: m_codes) {
            idx[j] = pos;
            memcpy(codes + pos, code->code, code->code_len);
            pos += code->code_len_align;
            j++;
        }

        gpuErrchk(hipMalloc((void**)&m_d_codes_buf, m_code_bytes));
        gpuErrchk(hipMalloc((void**)&m_d_codes_idx, m_codes.size() * sizeof(int)));
        gpuErrchk(hipMemcpy(m_d_codes_buf, codes, m_code_bytes,
                             hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(m_d_codes_idx, idx, m_codes.size() * sizeof(int),
                             hipMemcpyHostToDevice));

        delete[] codes;

        m_need_gpu_init = false;
    }

    int dim;
    for (dim = MIN_BLOCK_DIM; dim < MAX_BLOCK_DIM; dim += 32) {
        if (m_codes.size() <= m_grid_dim * dim) {
            break;
        }
    }

    m_block_dim = dim;
}

int
abpvm::skip_scheme(const char *sp)
{
    int i = 0;
    while (*sp !=':') {
        if (! schemechar[(unsigned char)*sp]) {
            return false;
        }
        sp++;
        i++;
    }

    sp++;
    i++;

    while (*sp == '/') {
        sp++;
        i++;
    }

    return i;
}

void
abpvm::match(std::vector<std::string> &result, const abpvm_query *query, int size)
{
    // TODO: check input
    init_gpu();

    int  *scheme_len;
    char *q_uri, *q_uri_lower;

    gpuErrchk(hipHostMalloc((void**)&scheme_len, MAX_QUERY_NUM * sizeof(scheme_len[0])));
    gpuErrchk(hipHostMalloc((void**)&q_uri, MAX_QUERY_LEN * MAX_QUERY_NUM));
    gpuErrchk(hipHostMalloc((void**)&q_uri_lower, MAX_QUERY_LEN * MAX_QUERY_NUM));

    int n = 0;

    for (int i = 0; i < size; i += MAX_QUERY_NUM) {
        int query_num = 0;
        for (query_num = 0; i + query_num < size &&
                            query_num < MAX_QUERY_NUM; query_num++) {
            int idx = i + query_num;
            int len = query[idx].get_len();
            const char *uri = query[idx].get_uri();
            const char *uri_lower = query[idx].get_uri_lower();

            scheme_len[query_num] = skip_scheme(uri_lower);

            len = (len < MAX_QUERY_LEN) ? len : MAX_QUERY_LEN;

            memcpy(q_uri + query_num * MAX_QUERY_LEN, uri, len);
            memcpy(q_uri_lower + query_num * MAX_QUERY_LEN, uri_lower, len);

            q_uri[query_num * MAX_QUERY_LEN + MAX_QUERY_LEN - 1] = '\0';
            q_uri_lower[query_num * MAX_QUERY_LEN + MAX_QUERY_LEN - 1] = '\0';

            n++;
        }

        gpuErrchk(hipMemcpy(m_d_query, q_uri, query_num * MAX_QUERY_LEN,
                             hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(m_d_query_lower, q_uri_lower, query_num * MAX_QUERY_LEN,
                             hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(m_d_scheme_len, scheme_len, query_num * sizeof(scheme_len[0]),
                             hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(m_d_result, m_result_init,
                             MAX_QUERY_NUM * MAX_RESULT * sizeof(m_result_init[0]),
                             hipMemcpyHostToDevice));

        gpu_match<<<m_grid_dim, m_block_dim>>>(m_d_codes_buf,
                                               m_d_codes_idx,
                                               m_codes.size(),
                                               m_d_scheme_len,
                                               m_d_query,
                                               m_d_query_lower,
                                               query_num,
                                               m_d_result);

        //hipDeviceSynchronize();
    }

    hipFree(scheme_len);
    hipFree(q_uri);
    hipFree(q_uri_lower);
}

bool
abpvm::vmrun(const char *pc, const char *sp)
{
    for (;;) {
        if (IS_OP_CHAR(*pc)) {
            if (*pc == CHAR_SEPARATOR) {
                if (! sepchar[(unsigned char)*sp]) {
                    return false;
                }
            } else {
                if (*pc != *sp) {
                    return false;
                }
            }
            sp++;
        } else if (IS_OP_MATCH(*pc)) {
            return true;
        } else if (IS_OP_SKIP_SCHEME(*pc)) {
            while (*sp !=':') {
                if (! schemechar[(unsigned char)*sp]) {
                    return false;
                }
                sp++;
            }

            sp++;

            while (*sp == '/') {
                sp++;
            }
        } else {
            // skip_to
            char c = 0x7f & *pc;
            if (c == CHAR_SEPARATOR) {
                while (! sepchar[(unsigned char)*sp]) {
                    if (*sp == '\0') {
                        return false;
                    }
                    sp++;
                }
            } else {
                while (c != *sp) {
                    if (*sp == '\0') {
                        return false;
                    }
                    sp++;
                }
            }
        }

        pc++;
    }

    // never reach here
    return true;
}

void
abpvm::print_asm()
{
    int total_inst = 0;
    int total_char = 0;
    int total_skip_to = 0;
    int total_skip_scheme = 0;
    int total_match = 0;

    for (auto &code: m_codes) {
        std::cout << "\"" << code->rule << "\"" << std::endl;

        abpvm_head *head = (abpvm_head*)code->code;
        char *inst = code->code + sizeof(abpvm_head);

        total_inst += head->num_inst;

        for (uint32_t j = 0; j < head->num_inst; j++, inst++) {
            if (IS_OP_CHAR(*inst)) {
                std::cout << "char ";
                total_char++;
                if (*inst == CHAR_HEAD) {
                    std::cout << "head" << std::endl;
                } else if (*inst == CHAR_TAIL) {
                    std::cout << "tail" << std::endl;
                } else if (*inst == CHAR_SEPARATOR) {
                    std::cout << "separator" << std::endl;
                } else {
                    std::cout << *inst << std::endl;
                }
            } else if (IS_OP_MATCH(*inst)) {
                std::cout << "match" << std::endl;
                total_match++;
            } else if (IS_OP_SKIP_SCHEME(*inst)) {
                std::cout << "skip_scheme" << std::endl;
                total_skip_scheme++;
            } else {
                char c = 0x7f & *inst;
                std::cout << "skip_to ";
                total_skip_to++;
                if (c == CHAR_HEAD) {
                    std::cout << "head" << std::endl;
                } else if (c == CHAR_TAIL) {
                    std::cout << "tail" << std::endl;
                } else if (c == CHAR_SEPARATOR) {
                    std::cout << "separator" << std::endl;
                } else {
                    std::cout << c << std::endl;
                }
            }
        }
        std::cout << std::endl;
    }

    std::cout << "#rule = " << m_codes.size()
              << "\n#instruction = " << total_inst
              << "\n#char = " << total_char
              << "\n#skip_to = " << total_skip_to
              << "\nskip_scheme = " << total_skip_scheme
              << "\nmatch = " << total_match
              << "\n" << std::endl;
}

void
abpvm::split(const std::string &str, const std::string &delim,
             std::vector<std::string> &ret)
{
    size_t current = 0, found, delimlen = delim.size();

    while((found = str.find(delim, current)) != std::string::npos) {
        ret.push_back(std::string(str, current, found - current));
        current = found + delimlen;
    }

    ret.push_back(std::string(str, current, str.size() - current));
}

void
abpvm::add_rule(const std::string &rule)
{
    std::vector<std::string> sp;
    std::string url_rule;
    std::shared_ptr<abpvm_code> code(new abpvm_code);
    uint32_t flags = 0;

    // do not add empty rules
    // do not add any comments
    if (rule.size() == 0 || rule.at(0) == '!') {
        return;
    }

    if (rule.find("##") != std::string::npos ||
        rule.find("#@#") != std::string::npos) {
        // TODO: element hide
        return;
    } else {
        // URL filter
        sp.clear();

        split(rule, "$", sp);

        if (sp.size() > 1) {
            std::vector<std::string> opts;

            split(sp[1], ",", opts);

            for (auto &opt: opts) {
                if (opt == "match-case") {
                    flags |= FLAG_MATCH_CASE;
                } else if (opt == "script") {
                    flags |= FLAG_SCRIPT;
                } else if (opt == "~script") {
                    flags |= FLAG_NOT_SCRIPT;
                } else if (opt == "image") {
                    flags |= FLAG_IMAGE;
                } else if (opt == "~image") {
                    flags |= FLAG_NOT_IMAGE;
                } else if (opt == "stylesheet") {
                    flags |= FLAG_STYLESHEET;
                } else if (opt == "~stylesheet") {
                    flags |= FLAG_NOT_STYLESHEET;
                } else if (opt == "object") {
                    flags |= FLAG_OBJECT;
                } else if (opt == "~object") {
                    flags |= FLAG_NOT_OBJECT;
                } else if (opt == "xmlhttprequest") {
                    flags |= FLAG_XMLHTTPREQUEST;
                } else if (opt == "~xmlhttprequest") {
                    flags |= FLAG_NOT_XMLHTTPREQUEST;
                } else if (opt == "object-subrequest") {
                    flags |= FLAG_OBJECT_SUBREQUEST;
                } else if (opt == "~object-subrequest") {
                    flags |= FLAG_NOT_OBJECT_SUBREQUEST;
                } else if (opt == "subdocument") {
                    flags |= FLAG_SUBDOCUMENT;
                } else if (opt == "~subdocument") {
                    flags |= FLAG_NOT_SUBDOCUMENT;
                } else if (opt == "document") {
                    flags |= FLAG_DOCUMENT;
                } else if (opt == "~document") {
                    flags |= FLAG_NOT_DOCUMENT;
                } else if (opt == "elemhide") {
                    flags |= FLAG_ELEMHIDE;
                } else if (opt == "~elemhide") {
                    flags |= FLAG_NOT_ELEMHIDE;
                } else if (opt == "other") {
                    flags |= FLAG_OTHER;
                } else if (opt == "~other") {
                    flags |= FLAG_NOT_OTHER;
                } else if (opt == "third-party") {
                    flags |= FLAG_THIRD_PARTY;
                } else if (opt == "~third-party") {
                    flags |= FLAG_NOT_THIRD_PARTY;
                } else if (opt == "collapse") {
                    flags |= FLAG_COLLAPSE;
                } else if (opt == "~collapse") {
                    flags |= FLAG_NOT_COLLAPSE;
                } else {
                    std::string s = opt.substr(0, 7); // domain=
                    if (s == "domain=") {
                        std::vector<std::string> sp2;
                        s = opt.substr(7);
                        split(s, "|", sp2);

                        for (auto &d: sp2) {
                            if (d.empty())
                                continue;

                            if (d.at(0) == '~') {
                                d.erase(0);
                                std::transform(d.begin(), d.end(),
                                               d.begin(), ::tolower);
                                abpvm_domain domain(d);
                                code->ex_domains.push_back(domain);
                            } else {
                                std::transform(d.begin(), d.end(),
                                               d.begin(), ::tolower);
                                abpvm_domain domain(d);
                                code->domains.push_back(domain);
                            }
                        }

                        flags |= FLAG_DOMAIN;
                    }
                }
            }
        }

        url_rule = sp[0];
        if (url_rule.size() >= 2 &&
            url_rule.at(0) == '@' && url_rule.at(1) == '@') {
            flags |= FLAG_NOT;
        }
    }

    // preprocess rule
    std::string result;

    std::regex re_multistar("\\*\\*+");
    std::regex re_tailstar("\\*$");
    std::regex re_headstar("^\\*");
    std::regex re_starbar("\\*\\|$");
    std::regex re_barstar("^\\|\\*");
    std::regex re_sepbar("\\^\\|$");

    url_rule = std::regex_replace(url_rule, re_multistar, "*");
    url_rule = std::regex_replace(url_rule, re_tailstar, "");
    url_rule = std::regex_replace(url_rule, re_headstar, "");
    url_rule = std::regex_replace(url_rule, re_starbar, "");
    url_rule = std::regex_replace(url_rule, re_barstar, "");
    url_rule = std::regex_replace(url_rule, re_sepbar, "^");

    int code_len;
    code->flags = flags;
    code->rule  = url_rule;
    code->code  = get_code(url_rule, flags, code_len);
    code->code_len  = code_len;
    code->original_rule = rule;

    int m = code_len % 4;
    code->code_len_align = code_len + ((m > 0) ? 4 - m : 0);

    if (code->code != nullptr) {
        m_codes.push_back(code);
        m_need_gpu_init = true;
        m_code_bytes += code->code_len_align;
    }
}

char *
abpvm::get_code(const std::string &rule, uint32_t flags, int &len)
{
    abpvm_head head;
    char inst[INST_MAX];
    const char *sp = rule.c_str();

    head.num_inst = 0;
    head.flags    = flags;

    if (sp[0] == '@' && sp[1] == '@') {
        sp += 2;
    }

    if (sp[0] == '|') {
        if (sp[1] == '|') {
            inst[0] = CHAR_HEAD;
            inst[1] = OP_SKIP_SCHEME;

            sp += 2;
            head.num_inst += 2;
        } else {
            inst[0] = CHAR_HEAD;

            sp++;
            head.num_inst++;
        }
    }

    while (*sp != '\0') {
        if (head.num_inst >= INST_MAX - 1) {
            // too many instructions
            std::ostringstream oss;
            oss << rule << ":\n"
                << "\ttoo many instructions (exceeded " << INST_MAX << ")";
            throw(abpvm_exception(oss.str()));
        }

        if (sp[0] == '*') {
            inst[head.num_inst] = OP_SKIP_TO;

            if (sp[1] == '^') {
                inst[head.num_inst] |= CHAR_SEPARATOR;
            } else {
                if (urlchar[(unsigned char)sp[1]]) {
                    if (flags & FLAG_MATCH_CASE) {
                        inst[head.num_inst] |= sp[1];
                    } else {
                        inst[head.num_inst] |= TO_LOWER(sp[1]);
                    }
                } else {
                    // invalid character
                    std::ostringstream oss;
                    oss << rule << ":\n"
                        << "\tinvalid character at " << &sp[1] - rule.c_str()
                        << " (" << sp[1] << ")";
                    throw(abpvm_exception(oss.str()));
                }
            }

            sp += 2;
        } else if (sp[0] == '^') {
            inst[head.num_inst] = CHAR_SEPARATOR;
            sp++;
        } else if (sp[0] == '|') {
            if (sp[1] == '\0') {
                inst[head.num_inst] = CHAR_TAIL;
            } else {
                // parse error
                std::ostringstream oss;
                oss << rule << ":\n"
                    << "\tinvalid character at " << &sp[0] - rule.c_str()
                    << " (" << sp[0] << ")";
                throw(abpvm_exception(oss.str()));
            }

            sp++;
        } else {
            if (urlchar[(unsigned char)sp[0]]) {
                inst[head.num_inst] = OP_CHAR;

                if (flags & FLAG_MATCH_CASE) {
                    inst[head.num_inst] = sp[0];
                } else {
                    inst[head.num_inst] = TO_LOWER(sp[0]);
                }
            } else {
                // invalide character
                std::ostringstream oss;
                oss << rule << ":\n"
                    << "\tinvalid character at " << &sp[0] - rule.c_str()
                    << " (" << sp[0] << ")";
                throw(abpvm_exception(oss.str()));
            }

            sp++;
        }

        head.num_inst++;
    }

    inst[head.num_inst] = OP_MATCH;
    head.num_inst++;

    if (head.num_inst > 0) {
        char *code = new char[sizeof(head) + sizeof(inst[0]) * head.num_inst];
        memcpy(code, &head, sizeof(head));
        memcpy(code + sizeof(head), inst, sizeof(inst[0]) * head.num_inst);

        len = head.num_inst + sizeof(head);
        return code;
    } else {
        return nullptr;
    }
}
