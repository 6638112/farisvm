#include "hip/hip_runtime.h"
#include "abpvm.hpp"

#include <algorithm>
#include <iostream>
#include <sstream>
#include <regex>

#include <stdio.h>

#define INST_MAX 4096

#define OP_CHAR        0x00
#define OP_SKIP_TO     0x80
#define OP_SKIP_SCHEME 0x83
#define OP_MATCH       0x84

#define CHAR_TAIL      0
#define CHAR_HEAD      1
#define CHAR_SEPARATOR 2

#define IS_OP_CHAR(INST) (!(0x80 & (INST)))
#define IS_OP_SKIP_SCHEME(INST) ((char)0x83 == (INST))
#define IS_OP_MATCH(INST) ((char)0x84 == (INST))

#define TO_LOWER(CH_) (('A' <= (CH_) && (CH_) <= 'Z') ? (CH_) + ('a' - 'A') : (CH_))
#define UNSIGNED(CH_) (int)(unsigned char)(CH_)

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void
gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
                file, line);
        if (abort) exit(code);
    }
}

// characters for URL by RFC 3986
int urlchar[256] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,
                    1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 1, 0, 1,
                    1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,
                    1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 1, 0, 1,
                    0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,
                    1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 1, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

// separators
int sepchar[256] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    1, 1, 1, 1, 1, 0, 1, 1, 1, 1, 1, 1, 1, 0, 0, 1,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1,
                    1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 0,
                    1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

int schemechar[256] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                       0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                       0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 1, 1, 0,
                       1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0,
                       0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,
                       1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0,
                       0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,
                       1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0,
                       0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                       0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                       0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                       0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                       0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                       0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                       0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                       0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

__constant__ int  d_urlchar[256];
__constant__ int  d_sepchar[256];
__constant__ int  d_schemechar[256];
__constant__ char d_query[1024 * 8];

__device__
void
cu_print_asm(char **codes, int num_codes)
{
    struct abpvm_head {
        uint32_t flags;
        uint32_t num_inst;
    };

    for (int i = 0; i < 100; i++) {
        abpvm_head *head = (abpvm_head*)codes[i];
        char *inst = codes[i] + sizeof(abpvm_head);

        for (uint32_t j = 0; j < head->num_inst; j++, inst++) {
            if (IS_OP_CHAR(*inst)) {
                printf("char ");
                if (*inst == CHAR_HEAD) {
                    printf("head\n");
                } else if (*inst == CHAR_TAIL) {
                    printf("tail\n");
                } else if (*inst == CHAR_SEPARATOR) {
                    printf("separator\n");
                } else {
                    printf("%c\n", *inst);
                }
            } else if (IS_OP_MATCH(*inst)) {
                printf("match\n");
            } else if (IS_OP_SKIP_SCHEME(*inst)) {
                printf("skip_scheme\n");
            } else {
                char c = 0x7f & *inst;
                printf("skip_to ");
                if (c == CHAR_HEAD) {
                    printf("head\n");
                } else if (c == CHAR_TAIL) {
                    printf("tail\n");
                } else if (c == CHAR_SEPARATOR) {
                    printf("separator\n");
                } else {
                    printf("%c\n", c);
                }
            }
        }

        printf("\n");
    }
}

__global__
void
cu_vmrun(char **codes, int num_codes)
{

};

abpvm_exception::abpvm_exception(const std::string msg) : m_msg(msg)
{

}

abpvm_exception::~abpvm_exception() throw()
{

}

const char*
abpvm_exception::what() const throw()
{
    return m_msg.c_str();
}

void
abpvm_query::set_uri(const std::string &uri)
{
    m_uri = uri;

    size_t colon = m_uri.find(":");
    if (colon == std::string::npos) {
        m_domain = "";
        return;
    }

    size_t begin = colon + 1;
    while (begin < m_uri.size() && m_uri.at(begin) == '/') {
        begin++;
    }

    if (begin >= m_uri.size()) {
        m_domain = "";
        return;
    }

    size_t end = begin + 1;
    while (end < m_uri.size() && m_uri.at(end) != '/') {
        end++;
    }

    m_domain = uri.substr(begin, end - begin);
}

abpvm::abpvm() : m_need_cu_init(true)
{
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_urlchar), urlchar, sizeof(urlchar)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_sepchar), sepchar, sizeof(sepchar)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_schemechar), schemechar, sizeof(schemechar)));
}

abpvm::~abpvm()
{
    for (auto &p: m_codes) {
        hipFree(p.d_code);
        delete p.code;
    }

    if (m_d_codes != nullptr) {
        hipFree(m_d_codes);
    }
}

void
abpvm::init_gpumem()
{
    if (m_need_cu_init) {
        if (m_d_codes != nullptr){
            hipFree(m_d_codes);
        }
        gpuErrchk(hipMalloc((void**)&m_d_codes,
                             m_codes.size() * sizeof(m_d_codes[0])));

        int num_codes = m_codes.size();

        for (int i = 0; i < num_codes; i++) {
            abpvm_head *head = (abpvm_head*)m_codes[i].code;
            uint32_t len = head->num_inst + sizeof(*head);
            gpuErrchk(hipMalloc((void**)&m_codes[i].d_code, len));
            gpuErrchk(hipMemcpy(m_codes[i].d_code, m_codes[i].code, len,
                                 hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(&m_d_codes[i], &m_codes[i].d_code, sizeof(char*),
                                 hipMemcpyHostToDevice));
        }

        m_need_cu_init = false;

        std::cout << "init rules" << std::endl;
    }
}

void
abpvm::match(std::vector<std::string> &result, const abpvm_query *query, int size)
{
    // TODO: check input

    init_gpumem();

    cu_vmrun<<<1, 1>>>(m_d_codes, m_codes.size());
    hipDeviceSynchronize();

    return;

    for (int i = 0; i < size; i++) {
        for (auto &code: m_codes) {
            abpvm_head *head = (abpvm_head*)code.code;
            char *pc = code.code + sizeof(*head);
            bool check_head = false;
            bool ret = false;

            if (*pc == CHAR_HEAD) {
                check_head = true;
                pc++;
            }

            const std::string &uri(query[i].get_uri());
            for (int j = 0; j < uri.size(); j++) {
                const char *sp = uri.c_str() + j;

                ret = vmrun(head, pc, sp);

                if (ret || check_head) {
                    break;
                }
            }

            if (ret) {
                // TODO: check options
                // check domains
                if (code.flags & FLAG_DOMAIN) {
                    const std::string &qd(query[i].get_domain());
                    std::string::const_iterator search_result;

                    for (auto &d: code.ex_domains) {
                        search_result = (*d.bmh)(qd.begin(), qd.end());
                        if (search_result == qd.end()) {
                            continue;
                        }
                    }

                    for (auto &d: code.domains) {
                        search_result = (*d.bmh)(qd.begin(), qd.end());
                        if (search_result != qd.end()) {
                            goto found;
                        }
                    }

                    continue;
                }
found:
                result.push_back(code.original_rule);
            }
        }
    }
}

bool
abpvm::vmrun(const abpvm_head *head, const char *pc, const char *sp)
{
    for (;;) {
        if (IS_OP_CHAR(*pc)) {
            if (*pc == CHAR_SEPARATOR) {
                if (! sepchar[(unsigned char)*sp]) {
                    return false;
                }
            } else {
                if (*pc != *sp) {
                    return false;
                }
            }
            sp++;
        } else if (IS_OP_MATCH(*pc)) {
            return true;
        } else if (IS_OP_SKIP_SCHEME(*pc)) {
            while (*sp !=':') {
                if (! schemechar[(unsigned char)*sp]) {
                    return false;
                }
                sp++;
            }

            sp++;

            while (*sp == '/') {
                sp++;
            }
        } else {
            // skip_to
            char c = 0x7f & *pc;
            if (c == CHAR_SEPARATOR) {
                while (! sepchar[(unsigned char)*sp]) {
                    if (*sp == '\0') {
                        return false;
                    }
                    sp++;
                }
            } else {
                while (c != *sp) {
                    if (*sp == '\0') {
                        return false;
                    }
                    sp++;
                }
            }
        }

        pc++;
    }

    // never reach here
    return true;
}

void
abpvm::print_asm()
{
    int total_inst = 0;
    int total_char = 0;
    int total_skip_to = 0;
    int total_skip_scheme = 0;
    int total_match = 0;

    for (auto &code: m_codes) {
        std::cout << "\"" << code.rule << "\"" << std::endl;

        abpvm_head *head = (abpvm_head*)code.code;
        char *inst = code.code + sizeof(abpvm_head);

        total_inst += head->num_inst;

        for (uint32_t j = 0; j < head->num_inst; j++, inst++) {
            if (IS_OP_CHAR(*inst)) {
                std::cout << "char ";
                total_char++;
                if (*inst == CHAR_HEAD) {
                    std::cout << "head" << std::endl;
                } else if (*inst == CHAR_TAIL) {
                    std::cout << "tail" << std::endl;
                } else if (*inst == CHAR_SEPARATOR) {
                    std::cout << "separator" << std::endl;
                } else {
                    std::cout << *inst << std::endl;
                }
            } else if (IS_OP_MATCH(*inst)) {
                std::cout << "match" << std::endl;
                total_match++;
            } else if (IS_OP_SKIP_SCHEME(*inst)) {
                std::cout << "skip_scheme" << std::endl;
                total_skip_scheme++;
            } else {
                char c = 0x7f & *inst;
                std::cout << "skip_to ";
                total_skip_to++;
                if (c == CHAR_HEAD) {
                    std::cout << "head" << std::endl;
                } else if (c == CHAR_TAIL) {
                    std::cout << "tail" << std::endl;
                } else if (c == CHAR_SEPARATOR) {
                    std::cout << "separator" << std::endl;
                } else {
                    std::cout << c << std::endl;
                }
            }
        }
        std::cout << std::endl;
    }

    std::cout << "#rule = " << m_codes.size()
              << "\n#instruction = " << total_inst
              << "\n#char = " << total_char
              << "\n#skip_to = " << total_skip_to
              << "\nskip_scheme = " << total_skip_scheme
              << "\nmatch = " << total_match
              << "\n" << std::endl;
}

void
abpvm::split(const std::string &str, const std::string &delim,
             std::vector<std::string> &ret)
{
    size_t current = 0, found, delimlen = delim.size();

    while((found = str.find(delim, current)) != std::string::npos) {
        ret.push_back(std::string(str, current, found - current));
        current = found + delimlen;
    }

    ret.push_back(std::string(str, current, str.size() - current));
}

void
abpvm::add_rule(const std::string &rule)
{
    std::vector<std::string> sp;
    std::string url_rule;
    abpvm_code code;
    uint32_t flags = 0;

    // do not add empty rules
    // do not add any comments
    if (rule.size() == 0 || rule.at(0) == '!') {
        return;
    }

    if (rule.find("##") != std::string::npos ||
        rule.find("#@#") != std::string::npos) {
        // TODO: element hide
        return;
    } else {
        // URL filter
        sp.clear();

        split(rule, "$", sp);

        if (sp.size() > 1) {
            std::vector<std::string> opts;

            split(sp[1], ",", opts);

            for (auto &opt: opts) {
                if (opt == "match-case") {
                    flags |= FLAG_MATCH_CASE;
                } else if (opt == "script") {
                    flags |= FLAG_SCRIPT;
                } else if (opt == "~script") {
                    flags |= FLAG_NOT_SCRIPT;
                } else if (opt == "image") {
                    flags |= FLAG_IMAGE;
                } else if (opt == "~image") {
                    flags |= FLAG_NOT_IMAGE;
                } else if (opt == "stylesheet") {
                    flags |= FLAG_STYLESHEET;
                } else if (opt == "~stylesheet") {
                    flags |= FLAG_NOT_STYLESHEET;
                } else if (opt == "object") {
                    flags |= FLAG_OBJECT;
                } else if (opt == "~object") {
                    flags |= FLAG_NOT_OBJECT;
                } else if (opt == "xmlhttprequest") {
                    flags |= FLAG_XMLHTTPREQUEST;
                } else if (opt == "~xmlhttprequest") {
                    flags |= FLAG_NOT_XMLHTTPREQUEST;
                } else if (opt == "object-subrequest") {
                    flags |= FLAG_OBJECT_SUBREQUEST;
                } else if (opt == "~object-subrequest") {
                    flags |= FLAG_NOT_OBJECT_SUBREQUEST;
                } else if (opt == "subdocument") {
                    flags |= FLAG_SUBDOCUMENT;
                } else if (opt == "~subdocument") {
                    flags |= FLAG_NOT_SUBDOCUMENT;
                } else if (opt == "document") {
                    flags |= FLAG_DOCUMENT;
                } else if (opt == "~document") {
                    flags |= FLAG_NOT_DOCUMENT;
                } else if (opt == "elemhide") {
                    flags |= FLAG_ELEMHIDE;
                } else if (opt == "~elemhide") {
                    flags |= FLAG_NOT_ELEMHIDE;
                } else if (opt == "other") {
                    flags |= FLAG_OTHER;
                } else if (opt == "~other") {
                    flags |= FLAG_NOT_OTHER;
                } else if (opt == "third-party") {
                    flags |= FLAG_THIRD_PARTY;
                } else if (opt == "~third-party") {
                    flags |= FLAG_NOT_THIRD_PARTY;
                } else if (opt == "collapse") {
                    flags |= FLAG_COLLAPSE;
                } else if (opt == "~collapse") {
                    flags |= FLAG_NOT_COLLAPSE;
                } else {
                    std::string s = opt.substr(0, 7); // domain=
                    if (s == "domain=") {
                        std::vector<std::string> sp2;
                        s = opt.substr(7);
                        split(s, "|", sp2);

                        for (auto &d: sp2) {
                            if (d.empty())
                                continue;

                            if (d.at(0) == '~') {
                                d.erase(0);
                                std::transform(d.begin(), d.end(),
                                               d.begin(), ::tolower);
                                abpvm_domain domain(d);
                                code.ex_domains.push_back(domain);
                            } else {
                                std::transform(d.begin(), d.end(),
                                               d.begin(), ::tolower);
                                abpvm_domain domain(d);
                                code.domains.push_back(domain);
                            }
                        }

                        flags |= FLAG_DOMAIN;
                    }
                }
            }
        }

        url_rule = sp[0];
        if (url_rule.size() >= 2 &&
            url_rule.at(0) == '@' && url_rule.at(1) == '@') {
            flags |= FLAG_NOT;
        }
    }

    // preprocess rule
    std::string result;

    std::regex re_multistar("\\*\\*+");
    std::regex re_tailstar("\\*$");
    std::regex re_headstar("^\\*");
    std::regex re_starbar("\\*\\|$");
    std::regex re_barstar("^\\|\\*");
    std::regex re_sepbar("\\^\\|$");

    url_rule = std::regex_replace(url_rule, re_multistar, "*");
    url_rule = std::regex_replace(url_rule, re_tailstar, "");
    url_rule = std::regex_replace(url_rule, re_headstar, "");
    url_rule = std::regex_replace(url_rule, re_starbar, "");
    url_rule = std::regex_replace(url_rule, re_barstar, "");
    url_rule = std::regex_replace(url_rule, re_sepbar, "^");

    code.flags = flags;
    code.rule  = url_rule;
    code.code  = get_code(url_rule, flags);

    code.original_rule = rule;

    if (code.code != nullptr)
        m_codes.push_back(code);

    m_need_cu_init = true;
}

char *
abpvm::get_code(const std::string &rule, uint32_t flags)
{
    abpvm_head head;
    char inst[INST_MAX];
    const char *sp = rule.c_str();

    head.num_inst = 0;
    head.flags    = flags;

    if (sp[0] == '@' && sp[1] == '@') {
        sp += 2;
    }

    if (sp[0] == '|') {
        if (sp[1] == '|') {
            inst[0] = CHAR_HEAD;
            inst[1] = OP_SKIP_SCHEME;

            sp += 2;
            head.num_inst += 2;
        } else {
            inst[0] = CHAR_HEAD;

            sp++;
            head.num_inst++;
        }
    }

    while (*sp != '\0') {
        if (head.num_inst >= INST_MAX - 1) {
            // too many instructions
            std::ostringstream oss;
            oss << rule << ":\n"
                << "\ttoo many instructions (exceeded " << INST_MAX << ")";
            throw(abpvm_exception(oss.str()));
        }

        if (sp[0] == '*') {
            inst[head.num_inst] = OP_SKIP_TO;

            if (sp[1] == '^') {
                inst[head.num_inst] |= CHAR_SEPARATOR;
            } else {
                if (urlchar[(unsigned char)sp[1]]) {
                    if (flags & FLAG_MATCH_CASE) {
                        inst[head.num_inst] |= sp[1];
                    } else {
                        inst[head.num_inst] |= TO_LOWER(sp[1]);
                    }
                } else {
                    // invalid character
                    std::ostringstream oss;
                    oss << rule << ":\n"
                        << "\tinvalid character at " << &sp[1] - rule.c_str()
                        << " (" << sp[1] << ")";
                    throw(abpvm_exception(oss.str()));
                }
            }

            sp += 2;
        } else if (sp[0] == '^'){
            inst[head.num_inst] = CHAR_SEPARATOR;
            sp++;
        } else if (sp[0] == '|') {
            if (sp[1] == '\0') {
                inst[head.num_inst] = CHAR_TAIL;
            } else {
                // parse error
                std::ostringstream oss;
                oss << rule << ":\n"
                    << "\tinvalid character at " << &sp[0] - rule.c_str()
                    << " (" << sp[0] << ")";
                throw(abpvm_exception(oss.str()));
            }

            sp++;
        } else {
            if (urlchar[(unsigned char)sp[0]]) {
                inst[head.num_inst] = OP_CHAR;

                if (flags & FLAG_MATCH_CASE) {
                    inst[head.num_inst] = sp[0];
                } else {
                    inst[head.num_inst] = TO_LOWER(sp[0]);
                }
            } else {
                // invalide character
                std::ostringstream oss;
                oss << rule << ":\n"
                    << "\tinvalid character at " << &sp[0] - rule.c_str()
                    << " (" << sp[0] << ")";
                throw(abpvm_exception(oss.str()));
            }

            sp++;
        }

        head.num_inst++;
    }

    inst[head.num_inst] = OP_MATCH;
    head.num_inst++;

    if (head.num_inst > 0) {
        char *code = new char[sizeof(head) + sizeof(inst[0]) * head.num_inst];

        memcpy(code, &head, sizeof(head));
        memcpy(code + sizeof(head), inst, sizeof(inst[0]) * head.num_inst);

        return code;
    } else {
        return nullptr;
    }
}
